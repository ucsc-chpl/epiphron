#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <random>
#include <list>
#include <hip/hip_runtime.h>
#include <device_atomic_functions.h>
#include <hipcub/hipcub.hpp>
//#include <cub/device/device_radix_sort.cuh>

using namespace std;
ofstream benchmarkData; 

// https://github.com/NVIDIA/cccl/blob/1cfe171ee948626668aa90a1922d744ff69b9ecd/cub/benchmarks/bench/radix_sort/keys.cu
// create policy
// attach policy to dispatch radix sort
// how to dispatch a dispatch_t
// bar graph, single value, other one line graph
float benchmark_cub_radixsort(int* d_keys_in, int* d_keys_out, int num_items) {

    void* d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;

    hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, num_items);
    hipMalloc(&d_temp_storage, temp_storage_bytes);

    hipEvent_t start, stop;

    float observed_rate = 0.0;

    for (int i = 1; i <= 3; i++) {
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, num_items);
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float kernelTime = 0.0;
        hipEventElapsedTime(&kernelTime, start, stop);
        observed_rate += (float(num_items) / float((kernelTime / 1000.0)));

    }
    observed_rate /= float(3);

    hipFree(d_keys_out);
    hipFree(d_temp_storage);

    return observed_rate;
}
int main(int argc, char *argv[]) {

    // Setup benchmark results file 
    benchmarkData.open("result.txt"); 
    if (!benchmarkData.is_open()) {
        cerr << "Failed to open the output file." << endl;
        return 1;
    }

    // number of elements
    for (int num_items = 1 << 20; num_items <= 1 << 28; num_items *= 2) {
        
        // Host input vector
        int* h_in;

        // Device input/output vector
        int *d_in, *d_out;

        //Size, in bytes, of input size
        size_t input_size = num_items * sizeof(int);

        // Allocate memory for vector on host
        h_in = (int *)malloc(input_size);

        // Allocate memory for vector on GPU
        hipMalloc(&d_in, input_size);
        hipMalloc(&d_out, input_size);

        // Initialize vectors on host
        for (int i = 0; i < num_items; i++) h_in[i] = rand() % 100; // seed this

        // Copy host vectors to device
        hipMemcpy(d_in, h_in, input_size, hipMemcpyHostToDevice);

        benchmarkData << "(" << input_size / (1024 * 1024) << ", ";

        // Run benchmark
        //float onesweep_res = benchmark_cub_onesweep(d_in, num_items);
        float radixsort_res = benchmark_cub_radixsort(d_in, d_out, num_items);

        benchmarkData << radixsort_res << ")" << endl;

        // Release device and host memory
        hipFree(d_in);
        free(h_in);
    }
    
    benchmarkData.close();

    return 0;

}
